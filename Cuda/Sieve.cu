#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

//Serial GPU sieve.
__global__ void sieve(int *nums, int n)
{
    int i,j;
    for(i = 2; i <= (int)sqrt((double)n); i++)
    {
        if(nums[i] > 0)
        {
            for(j = i+i; j <= n; j += i) nums[j] = 0;
        }
    }
}

//Serial CPU sieve.
void cpusieve(int *nums, int n)
{
    int i,j;
    for(i = 2; i <= (int)sqrt((double)n); i++)
    {
        if(nums[i] > 0)
        {
            for(j = i+i; j <= n; j += i) nums[j] = 0;
        }
    }
}

//Parallel GPU sieve.
__global__ void psieve(int *nums, int s, int n)
{
    int i;
    int bid = blockIdx.x + s; //current number
    int bidStart = bid + bid; //Where to start if prime.
 
    //If this number is prime (not marked composite)
    if(nums[bid] > 0)
    {
        int span = (n-bidStart)/bid; //The span of multiples to mark.
        int thit = span/blockDim.x + 1; //How many each thread will hit.
        int tid = bidStart + threadIdx.x*(thit+1)*bid; //The starting index of each thread.
 
        int bound = tid + bid*thit > n ? n : tid + bid*thit; //Last thread, might overshoot. Go til n.
        for(i = tid; i <= bound; i += bid) //Increment by number.
        {
            nums[i] = 0;    
        }
    }
}
int main(int argc, char **argv)
{
    int i, n, c, arg, nsqrt, blocks, s, t, *numbers, *dev_numbers,*params, *dev_params;
    clock_t start, end;
    n = atoi(argv[1]);
    nsqrt = (int)sqrt((double)n);
    arg = atoi(argv[2]);
 
    //Start the clock, and initialize array of numbers from 1 to n
    start = clock();
    numbers = (int *)malloc(sizeof(int)*(n+1));
    for(i = 0; i <= n; i++) numbers[i] = i;
 
    //Do serial CPU
    if(arg == 0)
    {
        cpusieve(numbers,n);
        end = clock();
    }
 
    //hipMalloc and Memcpy required vars for both kernals.
    else if(arg == 1 || arg == 2)
    {
        hipMalloc((void**)&dev_numbers,sizeof(int)*(n+1));
        hipMemcpy(dev_numbers,numbers,sizeof(int)*(n+1),hipMemcpyHostToDevice);
     
        //If it's the serial kernel, run it.
        if(arg == 2) sieve<<<1,1>>>(dev_numbers,n);
     
        //If its parallel, init other needed vars, and run it.
        else
        {
            s = 2;
            t = 2;
  
            //Keep pushing through completed stretches of numbers.
            while(s <= nsqrt)
            {
                blocks = t < nsqrt ? t - s + 1 : nsqrt - s + 1;
                psieve<<<blocks,128>>>(dev_numbers,s,n);
                s = t + 1;
                t *= t;
            }
        }
     
        //Copy to host, and end clock for kernel funcs.
        hipMemcpy(numbers,dev_numbers,sizeof(int)*(n+1),hipMemcpyDeviceToHost);
        end = clock();
    }
 
    //Print the numbers
    if(argc > 3)
    {
        c = 1;
        for(i = 0 ; i <= n; i++) 
        {
            if(c % 25 == 0) 
            {
                printf("\n");
                c = 1;
            }
            if(numbers[i] > 1) 
            {
                printf("%d ",numbers[i]);
                c++;
            }
        }
    }
    printf("\nTime: %lf\n",((double)(end-start))/CLOCKS_PER_SEC);
}
